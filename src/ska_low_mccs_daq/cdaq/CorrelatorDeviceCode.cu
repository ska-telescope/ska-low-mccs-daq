#include "hip/hip_runtime.h"
//
// Created by Alessio Magro on 04/05/2018.
//

#include "DeviceCode.h"

// ====================== cuFFT callbacks ==============================

// For below callbacks:
//   offset is the number of elements from start of data
//   dataOut is global output array pointer

// Load callback
__device__ hipfftComplex _expandInputPrecision(void *dataIn, size_t offset,
                                              void *callerInfo, void *sharedPtr)
{
    complex8_t value = ((complex8_t *) dataIn)[offset];
    return { (float) ((value.x) / 128.0),
             (float) ((value.y) / 128.0) };
}

// Store callback
__device__ void _quantizeOutput(void *dataOut, size_t offset, hipfftComplex element,
                                void *callerInfo, void *sharedPtr)
{
    // Read in observation parameters
    CudaObsParams *p = (CudaObsParams *) callerInfo;

    // Calculate index in output buffer where to store value
    int fft_number = offset / p -> nof_channels;
    int channel = offset % p -> nof_channels;
    int pol = fft_number % p -> nof_pols;
    int antenna = fft_number / p -> nof_pols;
    int index = p->nof_pols * (channel * p->nof_antennas + antenna) + pol;

    // Quantise value
    float factor = p->nof_channels / (sqrtf(p->nof_channels) * 2);
    complex8_t val;
    val.x = (int8_t) (element.x * factor);
    val.y = (int8_t) (element.y * factor);

    // Store quantised value
    *(((complex8_t *) dataOut) + index) = val;
}

// Pointer to load and store callback functions
__device__ hipfftCallbackLoadC d_loadCallbackPtr = _expandInputPrecision;
__device__ hipfftCallbackStoreC d_storeCallbackPtr = _quantizeOutput;

// Generate FFT plans
void generateFFTPlan(hipfftHandle *plan, hipStream_t *stream, CudaObsParams *params)
{
    // Create FFT plan
    CuFFTCheckResult(hipfftCreate(plan));

    // Create CUDA stream
    CudaSafeCall(hipStreamCreate(stream));

    // Get observation information
    int nof_channels = params -> nof_channels;
    int nof_antennas = params ->nof_antennas;
    int nof_pols = params -> nof_pols;

    // Create plan
    int inembed = 1, onembed = 1; // These are ignored for rank 1 FFTs
    size_t ws = 0;

    CuFFTCheckResult(hipfftMakePlanMany(*plan,
                                       1,              // rank
                                       &nof_channels,  // size of each dimension
                                       &inembed,       // inembed (storage dimensions of of input data in memory)
                                       (int) nof_antennas,   // istride (distance between two successive input elements in input array)
                                       1,              // idist (distance between the first element of two consecutive signals in a batch in input array)
                                       &onembed,       // onembef (storage dimensions of output data in memory)
                                       1,              // ostride (distance between two successive output elements in output array)
                                       nof_channels,   // odist (distance between the first element of two consecutive signals in a batch in output array
                                       HIPFFT_C2C,      // transform type
                                       nof_antennas * nof_pols, // batch size of transform
                                       &ws));          // pointer to the sizes of the work areas

    // Set CUDA stream for plan
    CuFFTCheckResult(hipfftSetStream(*plan, *stream));

    // Allocate observation params area on GPU and copy info
    CudaObsParams *d_cuda_obs_params;
    CudaSafeCall(hipMalloc((void **)&d_cuda_obs_params, sizeof(CudaObsParams)));
    CudaSafeCall(hipMemcpy(d_cuda_obs_params, params,
                            sizeof(CudaObsParams),
                            hipMemcpyHostToDevice));

    // Copy load callback symbol to GPU
    hipfftCallbackLoadR h_loadCallbackPtr;
    CudaSafeCall(hipMemcpyFromSymbol(&h_loadCallbackPtr,
                                      HIP_SYMBOL(d_loadCallbackPtr),
                                      sizeof(h_loadCallbackPtr)));

    // Copy store callback symbol to GPU
    hipfftCallbackStoreC h_storeCallbackPtr;
    CudaSafeCall(hipMemcpyFromSymbol(&h_storeCallbackPtr,
                                      HIP_SYMBOL(d_storeCallbackPtr),
                                      sizeof(h_storeCallbackPtr)));

    // Assign load callback
    CuFFTCheckResult(hipfftXtSetCallback(*plan, (void **) &h_loadCallbackPtr,
                                        HIPFFT_CB_LD_COMPLEX, nullptr));

    // Assign store callback
    CuFFTCheckResult(hipfftXtSetCallback(*plan, (void **) &h_storeCallbackPtr,
                                        HIPFFT_CB_ST_COMPLEX,
                                        (void **) &d_cuda_obs_params));
}